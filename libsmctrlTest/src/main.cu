#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "../libsmctrl/libsmctrl.h"

#include <chrono>

#define CUDALOOPS 5000
constexpr int TOTAL_TPCs = 19; // 19 TPCs in our 3060 Ti

// This macro determines on how many SMs our GPU kernel
// will be optimal on. Each TPC contains 2 SMs in our 3060Ti
// that means each partition can handle ~2048 parallel ops per cycle.
// So if you divide OPTIMAL_WIDTH/2048 = 8, meaning this kernel
// will be optimal on 8 SMs. Feel free to change this value.
#define OPTIMAL_WIDTH 16384

unsigned long long my_mask = 0;

__global__ void vecAdd_kernel(double* a, double* b, double* c, int n) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n) {
		for (int i = 0; i < CUDALOOPS; ++i) {
			c[id] = c[id] + a[id] + b[id];
			c[id] = c[id] + a[id];
			c[id] = c[id] + b[id];
		}
	}
}

void vecAdd_cuda(double* a, double* b, double* c, int n) {
	double* da, * db, * dc;
	size_t bytes = n * sizeof(double);
	hipMalloc(&da, bytes);
	hipMalloc(&db, bytes);
	hipMalloc(&dc, bytes);

	hipMemcpy(da, a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(db, b, bytes, hipMemcpyHostToDevice);

	int blockSize = 1024, gridSize;
	gridSize = (int)ceil((float)n / blockSize);

	hipStream_t myStream;
	hipStreamCreate(&myStream);

	libsmctrl_set_stream_mask(myStream, my_mask);

	vecAdd_kernel << <gridSize, blockSize, 0, myStream >> > (da, db, dc, n);

	hipMemcpy(dc, c, bytes, hipMemcpyDeviceToHost);
	hipStreamSynchronize(myStream);

	hipFree(da);
	hipFree(db);
	hipFree(dc);
}

int main(int argc, char* argv[]) {
	double* a, * b, * c;

	int n = OPTIMAL_WIDTH;
	size_t nBytes = n * sizeof(double);
	a = (double*)malloc(nBytes);
	b = (double*)malloc(nBytes);
	c = (double*)malloc(nBytes);

	for (int i = 0; i < n; ++i) {
		a[i] = sin(i) * sin(i);
		b[i] = cos(i) * cos(i);
	}

	// First, we do a warmup kernel launch, because the first kernel takes an eternity
	vecAdd_cuda(a, b, c, n);

	for (int i = 0; i < TOTAL_TPCs; ++i) {
		auto startTime = std::chrono::high_resolution_clock::now();
		vecAdd_cuda(a, b, c, n);
		auto endTime = std::chrono::high_resolution_clock::now();
		auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count();
		printf("%d Partitions: %ldms\n", TOTAL_TPCs - i, ms);
		my_mask <<= 1;
		my_mask |= 1;
	}
	return 0;
}
