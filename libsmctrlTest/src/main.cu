#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "../libsmctrl/libsmctrl.h"

#include <chrono>

#define CUDALOOPS 5000
#define TOTAL_TPCs 20

unsigned long long my_mask = 0;

__global__ void vecAdd_kernel(double* a, double* b, double* c, int n) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n) {
		for (int i = 0; i < CUDALOOPS; ++i) {
			c[id] = c[id] + a[id] + b[id];
			c[id] = c[id] + a[id];
			c[id] = c[id] + b[id];
		}
	}
}

void vecAdd_cuda(double* a, double* b, double* c, int n) {
	double* da, * db, * dc;
	size_t bytes = n * sizeof(double);
	hipMalloc(&da, bytes);
	hipMalloc(&db, bytes);
	hipMalloc(&dc, bytes);

	hipMemcpy(da, a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(db, b, bytes, hipMemcpyHostToDevice);

	int blockSize = 1024, gridSize;
	gridSize = (int)ceil((float)n / blockSize);

	hipStream_t myStream;
	hipStreamCreate(&myStream);

	libsmctrl_set_stream_mask(myStream, my_mask);

	vecAdd_kernel << <gridSize, blockSize, 0, myStream >> > (da, db, dc, n);

	hipMemcpy(dc, c, bytes, hipMemcpyDeviceToHost);
	hipStreamSynchronize(myStream);

	hipFree(da);
	hipFree(db);
	hipFree(dc);
}

int main(int argc, char* argv[]) {
	double* a, * b, * c;

	int n = 10000;
	size_t nBytes = n * sizeof(double);
	a = (double*)malloc(nBytes);
	b = (double*)malloc(nBytes);
	c = (double*)malloc(nBytes);

	for (int i = 0; i < n; ++i) {
		a[i] = sin(i) * sin(i);
		b[i] = cos(i) * cos(i);
	}

	for (int i = 0; i < TOTAL_TPCs; ++i) {
		auto startTime = std::chrono::high_resolution_clock::now();
		vecAdd_cuda(a, b, c, n);
		auto endTime = std::chrono::high_resolution_clock::now();
		auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime).count();
		printf("%d- %dms\n", i + 1, ms);
		my_mask <<= 1;
		my_mask |= 1;
	}
	return 0;
}